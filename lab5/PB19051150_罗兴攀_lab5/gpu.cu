
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 32 // 块大小

__global__ void gemm_baseline(float *A, float *B, float *C, int N)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x; 
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y; 

    float sum = 0.f;
    if (idx_x < N && idx_y < N)
    { 
        for (int kk = 0; kk < N; ++kk)
        {
            sum += A[idx_y * N + kk] * B[kk * N + idx_x];
        }
        C[idx_y * N + idx_x] = sum;
    }
}

__global__ void gemm_block(float *A, float *B, float *C, int N) {

  __shared__ float s_a[SIZE][SIZE];
  __shared__ float s_b[SIZE][SIZE];

  int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
  int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

  float sum = 0.0;
  for (int bk = 0; bk < N; bk += SIZE) {
    s_a[threadIdx.y][threadIdx.x] = A[idx_y * N + bk + threadIdx.x];
    s_b[threadIdx.y][threadIdx.x] = B[(bk + threadIdx.y) * N + idx_x];
    __syncthreads();

    for (int i = 0; i < SIZE; ++i) {
      sum += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
    }
    __syncthreads();
  }

  if (idx_x < N && idx_y < N) {
    C[idx_y * N + idx_x] = sum;
  }
}

int main()
{
    float *A, *B, *C;
    int N = 512; 

    A = new float[N * N](); 
    B = new float[N * N]();
    C = new float[N * N]();
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // 在主机(CPU)中产生数据
    for (int i = 0; i < N * N; ++i)
    {
        A[i] = static_cast<float>(rand()%10);
        B[i] = static_cast<float>(rand()%10);
    }

    // 将数据从主机(CPU)内存拷贝到设备(GPU)内存
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(SIZE, SIZE, 1);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y, 1);
    //dim3 gridSize(2, 2, 1);
    // 使用kernel计算C矩阵
    gemm_block<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize(); 

    // 将数据从设备(GPU)内存拷贝回主机(CPU)内存
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // 进行结果验证
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            float result = 0;
            for (int kk = 0; kk < N; ++kk)
            {
                result += A[i * N + kk] * B[kk * N + j];
            }
            if (C[i * N + j] != result)
            { 
                printf("error!\n");
                exit(-1);
            }
        }
    }

    // 释放内存
    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}